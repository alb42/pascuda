
#include <hip/hip_runtime.h>
__global__ 
void add(int *a, int *b, int *c, int *n)
{
  int index = blockIdx.x;
  if( index < *n)
  {
    c[index] = a[index] + b[index] ;
  }
}
